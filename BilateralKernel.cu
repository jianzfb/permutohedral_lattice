
#include <hip/hip_runtime.h>
// kernel_example.cu.cc
//#define GOOGLE_CUDA 1

#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "BilateralKernel.h"

#include "tensorflow/core/framework/op_kernel.h"
//#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "PermutohedralLatticeGPU.cu"

using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;


// Define the GPU implementation that launches the CUDA kernel.
template <typename T>
void ExampleFunctor<GPUDevice, T>::operator()(const GPUDevice& d,
                                              T *input,
                                              T *reference_image,
                                              int num_super_pixels,
                                              int n_spatial_dims,
                                              int *spatial_dims,
                                              int n_input_channels,
                                              int n_reference_channels,
                                              float theta_alpha,
                                              float theta_beta) {

    int pd = n_reference_channels + n_spatial_dims;
    int vd = n_input_channels + 1;
    int n = num_super_pixels;

    T* positions;
    cudaMalloc((void**)&(positions), n*pd*sizeof(float));

    compute_bilateral_kernel_gpu(reference_image,
                                 positions,
                                 num_super_pixels,
                                 n_reference_channels,
                                 n_spatial_dims,
                                 spatial_dims,
                                 theta_alpha,
                                 theta_beta);



    lattice_filter_gpu(input, positions, pd, vd, n);

    cudaFree(positions);
}

// Explicitly instantiate functors for the types of OpKernels registered.
template struct ExampleFunctor<GPUDevice, float>;
//template struct ExampleFunctor<GPUDevice, int32>;

#endif  // GOOGLE_CUDA